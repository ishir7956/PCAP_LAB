#include <stdio.h>
#include<hip/hip_runtime.h>

#define N 4 
#define BLOCK_SIZE 2

// Kernel to perform matrix multiplication
__global__ void matrixMultiply(int *A, int *B, int *C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;

    for (int i = 0; i < N; i++) {
        sum += A[row * N + i] * B[i * N + col];
    }

    C[row * N + col] = sum;
}

int main() {
    int A[N][N], B[N][N], C[N][N]; // Input and output matrices

    // Initialize matrices A and B

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            A[i][j] = i * N + j;
            B[i][j] = j * N + i;
        }
    }

    int *d_A, *d_B, *d_C; // Device matrices

    // Allocate memory on the GPU
    hipMalloc((void **)&d_A, N * N * sizeof(int));
    hipMalloc((void **)&d_B, N * N * sizeof(int));
    hipMalloc((void **)&d_C, N * N * sizeof(int));

    // Copy matrices A and B from host to device
    hipMemcpy(d_A, A, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(int), hipMemcpyHostToDevice);

    // Define block and grid dimensions
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(N / BLOCK_SIZE, N / BLOCK_SIZE);

    // Launch the kernel
    matrixMultiply<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    // Copy the result matrix C from device to host
    hipMemcpy(C, d_C, N * N * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Print the result matrix C
    printf("Matrix C:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", C[i][j]);
        }
        printf("\n");
    }

    return 0;
}
