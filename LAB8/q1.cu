#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 4 
#define NNZ 5 

__global__ void csrMatrixVectorMul(const int* row_ptr, const int* col_indices, const float* values, const float* x, float* y) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
   
    if (tid < N) {
        float dot_product = 0.0f;
        int row_start = row_ptr[tid];
        int row_end = row_ptr[tid + 1];
       
        for (int j = row_start; j < row_end; j++) {
            dot_product += values[j] * x[col_indices[j]];
        }
       
        y[tid] = dot_product;
    }
}

int main() {
    int row_ptr[N + 1] = {0, 2, 3, 4,5};
    int col_indices[NNZ] = {0, 2, 1, 3, 2};
    float values[NNZ] = {2.0, 3.0, 4.0, 1.0, 5.0};

    float x[N] = {1.0, 2.0, 3.0, 4.0};
    float y[N]; 
    int *d_row_ptr, *d_col_indices;
    float *d_values, *d_x, *d_y;

    hipMalloc((void**)&d_row_ptr, (N + 1) * sizeof(int));
    hipMalloc((void**)&d_col_indices, NNZ * sizeof(int));
    hipMalloc((void**)&d_values, NNZ * sizeof(float));
    hipMalloc((void**)&d_x, N * sizeof(float));
    hipMalloc((void**)&d_y, N * sizeof(float));

    hipMemcpy(d_row_ptr, row_ptr, (N + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_indices, col_indices, NNZ * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values, values, NNZ * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 grid(1, 1, 1);
    dim3 block(N, 1, 1);

    csrMatrixVectorMul<<<grid, block>>>(d_row_ptr, d_col_indices, d_values, d_x, d_y);

    hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    printf("Result vector:\n");
    for (int i = 0; i < N; i++) {
        printf("%.2f ", y[i]);
    }
    printf("\n");

    hipFree(d_row_ptr);
    hipFree(d_col_indices);
    hipFree(d_values);
    hipFree(d_x);
    hipFree(d_y);

    return 0;
}



